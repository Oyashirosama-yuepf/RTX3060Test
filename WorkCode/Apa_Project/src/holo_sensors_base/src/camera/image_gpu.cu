#include "hip/hip_runtime.h"
/*
 * Copyright (C) HoloMatic Technology(Beijing) Co., Ltd. - All Rights Reserved
 * Unauthorized copying of this file, via any medium is strictly prohibited
 * Proprietary and confidential
 */

/**
 * @file image_gpu.cu.
 * @brief Image Encoder.
 * @author lanpade(yangpeiwen@holomatic.com).
 * @date 2020-07-31.
 */
#include <holo/sensors/camera/image_gpu.h>

inline __device__ __host__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}

/* From RGB to YUV

   Y = 0.299R + 0.587G + 0.114B
   U = 0.492 (B-Y)
   V = 0.877 (R-Y)

   It can also be represented as:

   Y =  0.299R + 0.587G + 0.114B
   U = -0.147R - 0.289G + 0.436B
   V =  0.615R - 0.515G - 0.100B

   From YUV to RGB

   R = Y + 1.140V
   G = Y - 0.395U - 0.581V
   B = Y + 2.032U
 */

struct __align__(2) uchar6
{
    uchar3 a0, a1;
};
static __host__ __device__ __forceinline__ uchar6 make_uchar6(uchar3 a0, uchar3 a1)
{
    uchar6 val = {a0, a1};
    return val;
}

__global__ void yuyvToRgb(uint8_t* yuyv, uint8_t* rgb, int srcAlignedWidth, int dstAlignedWidth, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= srcAlignedWidth || y >= height)
        return;

    const uchar4 macroPx = ((uchar4*)yuyv)[y * width / 2 + x];  // = src[y * srcAlignedWidth + x];

    // Y0 is the brightness of pixel 0, Y1 the brightness of pixel 1.
    // U0 and V0 is the color of both pixels.
    // UYVY [ U0 | Y0 | V0 | Y1 ]
    // YUYV [ Y0 | U0 | Y1 | V0 ]
    const float y0 = macroPx.x;
    const float y1 = macroPx.z;
    const float u  = (macroPx.y) - 128.0f;
    const float v  = (macroPx.w) - 128.0f;

    const float3 px0 = make_float3(y0 + 1.4065f * v, y0 - 0.3455f * u - 0.7169f * v, y0 + 1.7790f * u);

    const float3 px1 = make_float3(y1 + 1.4065f * v, y1 - 0.3455f * u - 0.7169f * v, y1 + 1.7790f * u);

    ((uchar6*)rgb)[y * width / 2 + x] =
        make_uchar6(make_uchar3(clamp(px0.x, 0.0f, 255.0f), clamp(px0.y, 0.0f, 255.0f), clamp(px0.z, 0.0f, 255.0f)),
                    make_uchar3(clamp(px1.x, 0.0f, 255.0f), clamp(px1.y, 0.0f, 255.0f), clamp(px1.z, 0.0f, 255.0f)));
}

__global__ void yuyvToYuv(uint8_t* yuyv, uint8_t* yuv, int srcAlignedWidth, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= srcAlignedWidth || y >= height)
        return;

    const uchar4 macroPx = ((uchar4*)yuyv)[y * width / 2 + x];  // = src[y * srcAlignedWidth + x];

    // Y0 is the brightness of pixel 0, Y1 the brightness of pixel 1.
    // U0 and V0 is the color of both pixels.
    // UYVY [ U0 | Y0 | V0 | Y1 ]
    // YUYV [ Y0 | U0 | Y1 | V0 ]
    const float y0 = macroPx.x;
    const float y1 = macroPx.z;
    const float u  = (macroPx.y);
    const float v  = (macroPx.w);

    ((uchar2*)yuv)[y * width / 2 + x] = make_uchar2(y0, y1);

    if (y % 2 == 0)
    {
        ((uchar1*)yuv)[width * height + y * width / 4 + x]                        = make_uchar1(u);
        ((uchar1*)yuv)[width * height + (width * height) / 4 + y * width / 4 + x] = make_uchar1(v);
    }
}

__global__ void rgbToYuv(uint8_t* rgb, uint8_t* yuv, int srcAlignedWidth, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= srcAlignedWidth || y >= height)
        return;

    const uchar6 macroPx = ((uchar6*)rgb)[y * width / 2 + x];

    const float r0 = macroPx.a0.x;
    const float g0 = macroPx.a0.y;
    const float b0 = macroPx.a0.z;
    const float r1 = macroPx.a1.x;
    const float g1 = macroPx.a1.y;
    const float b1 = macroPx.a1.z;

    float y0 = 0.257 * r0 + 0.504 * g0 + 0.098 * b0 + 16;
    float y1 = 0.257 * r1 + 0.504 * g1 + 0.098 * b1 + 16;

    ((uchar2*)yuv)[y * width / 2 + x] = make_uchar2(y0, y1);

    if (y % 2 == 0)
    {
        float u                                            = -0.148 * r0 - 0.291 * g0 + 0.439 * b0 + 128;
        float v                                            = 0.439 * r0 - 0.368 * g0 - 0.071 * b0 + 128;
        ((uchar1*)yuv)[width * height + y * width / 4 + x] = make_uchar1(u);
        ((uchar1*)yuv)[width * height + (width * height) / 4 + y * width / 4 + x] = make_uchar1(v);
    }
}

__global__ void yuvToRgb(uint8_t* yuv, uint8_t* rgb, int srcAlignedWidth, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= srcAlignedWidth || y >= height)
        return;

    const uchar2 macroPx = ((uchar2*)yuv)[y * width / 2 + x];

    float y0 = macroPx.x;
    float y1 = macroPx.y;
    int   y_id;

    if (y % 2 == 0)
    {
        y_id = y;
    }
    else
    {
        y_id = y - 1;
    }

    const uchar2 tt = make_uchar2(((uchar1*)yuv)[width * height + y_id * width / 4 + x].x,
                                  ((uchar1*)yuv)[width * height + (width * height) / 4 + y_id * width / 4 + x].x);
    // float u = ((uchar1*)yuv)[width * height + y_id * width/4 + x];// -128.0f;
    // uchar1 v = ((uchar1*)yuv)[width * height + (width*height)/4 + y_id * width/4 + x];

    // float u = tt.x - 128.0f;
    float u = clamp(((uchar1*)yuv)[width * height + y_id * width / 4 + x].x - 128.0f, 0.0f, 255.0f);
    float v =
        clamp(((uchar1*)yuv)[width * height + (width * height) / 4 + y_id * width / 4 + x].x - 128.0f, 0.0f, 255.0f);
    // float uu = (float) - 128.0f;
    // const float v =  tt.y - 128.0f;

    const float3 px0 = make_float3(y0 + 1.4065f * v, y0 - 0.3455f * u - 0.7169f * v, y0 + 1.7790f * u);

    const float3 px1 = make_float3(y1 + 1.4065f * v, y1 - 0.3455f * u - 0.7169f * v, y1 + 1.7790f * u);

    ((uchar6*)rgb)[y * width / 2 + x] =
        make_uchar6(make_uchar3(clamp(px0.x, 0.0f, 255.0f), clamp(px0.y, 0.0f, 255.0f), clamp(px0.z, 0.0f, 255.0f)),
                    make_uchar3(clamp(px1.x, 0.0f, 255.0f), clamp(px1.y, 0.0f, 255.0f), clamp(px1.z, 0.0f, 255.0f)));
}

void CudaYUYVToRGB(const int32_t cols, const int32_t rows, uint8_t* cuda_yuyv, uint8_t* cuda_rgb, uint8_t* yuyv,
                   uint8_t* rgbb_image)
{
    hipError_t cuda_status;

    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess)
    {
        printf("Device does not support cuda\n");
        return;
    }

    cuda_status = hipMemcpy(cuda_yuyv, yuyv, cols * rows * 2, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data to gpu\n");
        return;  // cuda_status;
    }

    dim3      block(8, 8);
    dim3      grid(cols / 2 / 8, rows / 8);
    const int srcAlignedWidth =
        cols * rows * 2 / sizeof(uchar4);  // normally would be uchar2, but we're doubling up pixels
    const int dstAlignedWidth = cols * rows * 3 / sizeof(uchar6);  // normally would be uchar4 ^^^

    // printf("yuyvToRgba %zu %zu %i %i %i %i %i\n", width, height, (int)formatUYVY, srcAlignedWidth, dstAlignedWidth,
    // grid.x, grid.y);

    yuyvToRgb<<<grid, block>>>(cuda_yuyv, cuda_rgb, srcAlignedWidth, dstAlignedWidth, cols, rows);

    cuda_status = hipGetLastError();

    if (cuda_status != hipSuccess)
    {
        printf("Yuyv to rgb error\n");
        return;
    }

    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess)
    {
        printf("sychronization error\n");
        return;
    }

    cuda_status = hipMemcpy(rgbb_image, cuda_rgb, cols * rows * 3, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data from gpu\n");
        return;
    }

    return;
}

void CudaYUYVToYUV(const int32_t cols, const int32_t rows, uint8_t* cuda_yuyv, uint8_t* cuda_yuv, uint8_t* yuyv,
                   uint8_t* yuv)
{
    hipError_t cuda_status;

    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess)
    {
        printf("Device does not support cuda\n");
        return;
    }

    cuda_status = hipMemcpy(cuda_yuyv, yuyv, cols * rows * 2, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data to gpu\n");
        return;  // cuda_status;
    }

    dim3      block(8, 8);
    dim3      grid(cols / 2 / 8, rows / 8);
    const int srcAlignedWidth = cols * rows * 2 / sizeof(uchar4);

    yuyvToYuv<<<grid, block>>>(cuda_yuyv, cuda_yuv, srcAlignedWidth, cols, rows);

    cuda_status = hipGetLastError();

    if (cuda_status != hipSuccess)
    {
        printf("Yuyv to rgb error\n");
        return;
    }

    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess)
    {
        printf("sychronization error\n");
        return;
    }

    cuda_status = hipMemcpy(yuv, cuda_yuv, cols * rows * 1.5, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data from gpu\n");
        return;
    }

    return;
}

void CudaRGBToYUV(const int32_t cols, const int32_t rows, uint8_t* cuda_rgb, uint8_t* cuda_yuv, const uint8_t* rgb,
                  uint8_t* yuv)
{
    hipError_t cuda_status;

    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess)
    {
        printf("Device does not support cuda\n");
        return;
    }

    cuda_status = hipMemcpy(cuda_rgb, rgb, cols * rows * 3, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data to gpu\n");
        return;  // cuda_status;
    }

    dim3      block(8, 8);
    dim3      grid(cols / 2 / 8, rows / 8);
    const int srcAlignedWidth = cols * rows * 2 / sizeof(uchar4);

    rgbToYuv<<<grid, block>>>(cuda_rgb, cuda_yuv, srcAlignedWidth, cols, rows);

    cuda_status = hipGetLastError();

    if (cuda_status != hipSuccess)
    {
        printf("rgb to yuv error\n");
        return;
    }

    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess)
    {
        printf("sychronization error\n");
        return;
    }

    cuda_status = hipMemcpy(yuv, cuda_yuv, cols * rows * 1.5, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data from gpu\n");
        return;
    }

    return;
}

void CudaRGBToYUV(const int32_t cols, const int32_t rows, uint8_t* cuda_rgb, uint8_t* cuda_yuv, const uint8_t* rgb,
                  uint8_t* y, uint8_t* u, uint8_t* v)
{
    hipError_t cuda_status;

    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess)
    {
        printf("Device does not support cuda\n");
        return;
    }

    cuda_status = hipMemcpy(cuda_rgb, rgb, cols * rows * 3, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data to gpu\n");
        return;  // cuda_status;
    }

    dim3      block(8, 8);
    dim3      grid(cols / 2 / 8, rows / 8);
    const int srcAlignedWidth = cols * rows * 2 / sizeof(uchar4);

    rgbToYuv<<<grid, block>>>(cuda_rgb, cuda_yuv, srcAlignedWidth, cols, rows);

    cuda_status = hipGetLastError();

    if (cuda_status != hipSuccess)
    {
        printf("rgb to yuv error\n");
        return;
    }

    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess)
    {
        printf("sychronization error\n");
        return;
    }

    uint32_t offset = 0u;
    cuda_status     = hipMemcpy(y, cuda_yuv + offset, cols * rows, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data from gpu\n");
        return;
    }
    offset += cols * rows;
    cuda_status = hipMemcpy(u, cuda_yuv + offset, cols * rows / 4, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data from gpu\n");
        return;
    }
    offset += cols * rows / 4;
    cuda_status = hipMemcpy(v, cuda_yuv + offset, cols * rows / 4, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data from gpu\n");
        return;
    }
}

void CudaYUVToRGB(const int32_t cols, const int32_t rows, uint8_t* cuda_yuv, uint8_t* cuda_rgb, uint8_t* yuv,
                  uint8_t* rgb)
{
    hipError_t cuda_status;

    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess)
    {
        printf("Device does not support cuda\n");
        return;
    }

    cuda_status = hipMemcpy(cuda_yuv, yuv, cols * rows * 1.5, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data to gpu\n");
        return;  // cuda_status;
    }

    dim3      block(8, 8);
    dim3      grid(cols / 2 / 8, rows / 8);
    const int srcAlignedWidth = cols * rows * 2 / sizeof(uchar4);

    yuvToRgb<<<grid, block>>>(cuda_yuv, cuda_rgb, srcAlignedWidth, cols, rows);

    cuda_status = hipGetLastError();

    if (cuda_status != hipSuccess)
    {
        printf("Yuv to rgb error\n");
        return;
    }

    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess)
    {
        printf("sychronization error\n");
        return;
    }

    cuda_status = hipMemcpy(rgb, cuda_rgb, cols * rows * 3, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess)
    {
        printf("Could not copy data from gpu\n");
        return;
    }

    return;
}
